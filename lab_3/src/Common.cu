#include "Common.h"

#include <iostream>

float CudaTimer::avgElapsedTime = 0.f;

CudaTimer::CudaTimer(bool quite) 
    : q(quite)
{
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

CudaTimer::~CudaTimer() 
{
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    if(avgElapsedTime != 0)
    {
        avgElapsedTime += elapsedTime;
        avgElapsedTime = avgElapsedTime / 2.f;
    }
    else avgElapsedTime += elapsedTime;
    
    if(!q) printf("Время выполнения: %.5f мс\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}