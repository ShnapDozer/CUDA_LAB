#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#define STB_IMAGE_IMPLEMENTATION   
#include "../stb-master/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../stb-master/stb_image_write.h"
#include <stdio.h>

__global__ void transformKernel(float* output,
    hipTextureObject_t texObj,
    int width, int height,
    float theta)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

    output[y * width + x] = tex2D<float>(texObj, tu, tv);
}

char* toBinary(int n, int len)
{
    char* binary = (char*)malloc(sizeof(char) * len);
    int k = 0;
    for (unsigned i = (1 << len - 1); i > 0; i = i / 2) {
        binary[k++] = (n & i) ? '1' : '0';
    }
    binary[k] = '\0';
    return binary;
}

void print_binary(unsigned char* n)
{
    int len = 8;
    char* binary = toBinary(n[0], len);
    printf("The binary representation of %d is %s\n", n[0], binary);
}

void print_binary(float* n)
{
    int len = 32;
    char* binary = toBinary(n[0], len);
    printf("The binary representation of %d is %s\n", n[0], binary);
}

int main()
{
    int height = 320;
    int width = 213;
    int texChannels;
    float angle = 45;

    stbi_uc* pixels = stbi_load("./cat.bmp", &width, &height, &texChannels, STBI_grey);
    if(!pixels)  printf("not loaded\n");
    printf("loaded\n");

    float* h_data = (float*)malloc(sizeof(float) * width * height);
    for (int i = 0; i < height * width; ++i)
        h_data[i] = (pixels[i] & 0xff);

    print_binary(pixels);
    print_binary(h_data);

    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    float* output;
    hipMalloc(&output, width * height * sizeof(float));


    const size_t spitch = width * sizeof(float);
    hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float),
        height, hipMemcpyHostToDevice);

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x, (height + threadsperBlock.y - 1) / threadsperBlock.y);
    transformKernel <<< numBlocks, threadsperBlock >>>(output, texObj, width, height, angle);
    
    hipMemcpy(h_data, output, width * height * sizeof(float), hipMemcpyDeviceToHost);

    unsigned char* h_data_char = (unsigned char*)malloc(sizeof(unsigned char) * width * height);
  
    print_binary(h_data);

    for (int i = 0; i < height * width; ++i)
    {
        h_data_char[i] = (unsigned char)h_data[i];
    }
        
    print_binary(h_data_char);

    stbi_write_jpg("./cat-out.jpg", width, height, 1, h_data_char, 100);
    printf("write\n");
    
    hipDestroyTextureObject(texObj);

    hipFreeArray(cuArray);
    hipFree(output);
    stbi_image_free(pixels);
    free(h_data);

    return 0;
}